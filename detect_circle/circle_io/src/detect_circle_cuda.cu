#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdbool.h>
#include <math.h>
#include <math_functions.h>
#include <time.h>
#include <sys/time.h>
#include <vector>
#include <boost/optional.hpp>
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/sort.h>
#include "../include/detect_circle_cuda_p.hpp"

boost::optional<position> detect_circle_cuda(const std::vector<float>& host_ranges, const int lrf_num, const float angle_min, const float angle_increment, const hough_param_str *hough_param, const int thr2, const float p, const float q, const float rad, const float rad_err1, const float rad_err2, const float rad_err3, const float allow_err1, const float allow_err2, const int warp, const int limit_count){
  /*
    struct timeval start,end;
    時間計測用
    以下使い方
    gettimeofday(&start,NULL);
    hogehoge;
    gettimeofday(&end,NULL);
    printf("time:%d[ns]\n", end.tv_usec - start.tv_usec);

    float elapsed_time_ms=0.0f;
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
    hoge<<<,>>>()
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time_ms,start,stop);
    printf("time:%8.2f ms\n",elapsed_time_ms);
    cudaEventDestory(start);
    cudaEventDestory(end);
  */

  boost::optional<position> position_data;

  if (lrf_num > 1024)
  {
    std::cout << "too many number to see." << std::endl;
    return position_data;
  }
  //gpuの可能並列化数を超えていたらreturn

  const size_t ranges_num = host_ranges.size();
  if (ranges_num < lrf_num)
  {
    // LRFの距離データの数が少なすぎた時.
    std::cout << "few number of lrf data." << std::endl;
    return position_data;
  }

  thrust::device_vector<float> device_ranges(host_ranges.size());
  // デバイス用の距離データ格納用のvector.
  thrust::device_vector<float> device_xy_data(2 * lrf_num);
  // デバイス用のx-yデータ格納用のvector.
  thrust::host_vector<float> host_xy_data(2 * lrf_num);
  // ホスト用のx-yデータ格納用のvector.
  thrust::copy(host_ranges.begin(), host_ranges.end(), device_ranges.begin());
  // デバイスに距離情報をコピー.

  dim3 block_0((int)floor((lrf_num+warp-1)/warp),1,1);
  dim3 thread_0(warp,1,1);
  // 並列化の大きさを設定.

  const int lrf_offset = (ranges_num - lrf_num) / 2;
  // lrfデータの中央部分のみを見るため,そのオフセットを指定.

  make_xy_data<<<block_0,thread_0>>>(
    (float *)thrust::raw_pointer_cast(device_ranges.data()),
    (float *)thrust::raw_pointer_cast(device_xy_data.data()),
    lrf_offset, lrf_num, angle_min, angle_increment);
  // x-yデータを作成.

  thrust::copy(device_xy_data.begin(), device_xy_data.end(), host_xy_data.begin());
  // ホストにx-yデータをコピー.

  const float x_range = hough_param->x_num / 2 * hough_param->x_wid;
  const float y_range = hough_param->y_num / 2 * hough_param->y_wid;
  // ハフ変換時に見る範囲.

  thrust::device_vector<int> device_hough_list(hough_param->x_num * hough_param->y_num, 0);
  // デバイス用のハフ変換データ格納用のvector.

  dim3 blocks_1(lrf_num, 1, 1);
  dim3 threads_1(hough_param->x_num,1,1);
  // 並列化.

  make_hough_graph<<<blocks_1,threads_1>>>((float *)thrust::raw_pointer_cast(device_xy_data.data()),
  (int *)thrust::raw_pointer_cast(device_hough_list.data()),
  p, q,
  hough_param->x_num, hough_param->y_num,
  hough_param->x_wid, hough_param->y_wid,
  x_range, y_range, rad);

  // print_hough_gragh(device_hough_list, x_num, y_num);
  // ハフ変換グラフを表示.

  thrust::device_vector<float> device_circle(2 * hough_param->x_num * hough_param->y_num);
  // デバイス用の検出円格納用のvector.
  thrust::host_vector<float> host_circle(2 * hough_param->x_num * hough_param->y_num);
  // ホスト用の検出円格納用のvector.

  thrust::device_vector<int> device_head(1);
  // デバイス用の検出円数格納用のvector.
  thrust::host_vector<int> host_head(1);
  // ホスト用の検出円数格納用のvector.

  host_head[0] = 0;

  thrust::copy(host_head.begin(), host_head.end(), device_head.begin());
  // デバイスに検出円数をコピー.

  dim3 blocks_2(hough_param->x_num, 1, 1);
  dim3 threads_2(hough_param->y_num, 1,1);

  find_circle<<<blocks_2,threads_2>>>((int *)thrust::raw_pointer_cast(device_hough_list.data()),
  (float *)thrust::raw_pointer_cast(device_circle.data()),
  (int *)thrust::raw_pointer_cast(device_head.data()),
  hough_param->near_x, hough_param->near_y,
  hough_param->x_num, hough_param->y_num,
  hough_param->thr, hough_param->weight);
  // 円検出.

  thrust::copy(device_circle.begin(), device_circle.end(), host_circle.begin());
  // ホストにx-yデータをコピー.
  thrust::copy(device_head.begin(), device_head.end(), host_head.begin());
  // ホストにx-yデータをコピー.


  float x_esti, y_esti;
  if(host_head[0] != 0){
    const float x_b = p, y_b = q;
    // x-yの中心位置(オフセット).
    x_esti = (float)(host_circle[0] - hough_param->x_num / 2) * hough_param->x_wid + x_b;
    y_esti = (float)(host_circle[1] - hough_param->y_num / 2) * hough_param->y_wid + y_b;
    // 検出したx-yデータ.

    float diff = (x_b-x_esti) * (x_b-x_esti) + (y_b-y_esti) * (y_b-y_esti);
    // 中心位置からのズレ.

    for (int i = 1; i < host_head[0]; i++){
      const float now_x = (float)(host_circle[2 * i] - hough_param->x_num / 2) * hough_param->x_wid + x_b;
      const float now_y = (float)(host_circle[2 * i + 1] - hough_param->y_num / 2) * hough_param->y_wid + y_b;
      const float now_diff = (x_b-now_x) * (x_b-now_x) + (y_b-now_y) * (y_b-now_y);
      if (diff > now_diff)
      {
        x_esti = now_x;
	      y_esti = now_y;
	      diff = now_diff;
      }
    }
  }
  // 予想の位置に一番近いものを選んでいる.

  /*
  if (host_head[0] != 0){
    printf("%d\n",host_head[0]);
    *p = x_esti;
    *q = y_esti;
    return position_data;
  }
  */

  if (host_head[0] != 0){
    // ここから2回目の処理.
    std::vector<float> select_datas;
    // 選別したlrfのデータを保管.
    boost::optional<position> calc_position;
    // 計算した位置の格納.

    select_data(host_xy_data, device_xy_data, &select_datas, lrf_num, x_esti, y_esti, rad, rad_err1, warp);
    //lrfデータを選別

    calc_position = mle(select_datas, rad, x_esti, y_esti, allow_err1, limit_count);
    //最尤推定を実行
    if(calc_position)
    {
      x_esti = calc_position->x;
      y_esti = calc_position->y;
    }
    else
    {
      return position_data;
    }

    select_data(host_xy_data, device_xy_data, &select_datas, lrf_num, x_esti, y_esti, rad, rad_err2, warp);
    //lrfデータを選別

    calc_position = mle(select_datas, rad, x_esti, y_esti, allow_err2, limit_count);
    //二回目の最尤推定を実行
    if(calc_position)
    {
      x_esti = calc_position->x;
      y_esti = calc_position->y;
    }
    else
    {
      return position_data;
    }

    select_data(host_xy_data, device_xy_data, &select_datas, lrf_num, x_esti, y_esti, rad, rad_err3, warp);
    //printf("select:%d\n",select_data_num);

    if(select_datas.size() > (size_t)thr2)
    {
      position tmp_position = {x_esti, y_esti};
      position_data = tmp_position;
    }
  }

  return position_data;
}

__global__ void make_xy_data(const float *device_ranges, float *device_xy_data, const int lrf_offset, const int lrf_num, const float angle_min, const float angle_increment)
{
  // 距離データをx-yデータに変換する.

  const int num = blockIdx.x * blockDim.x + threadIdx.x;
  // 見るべき配列の値.

  if (lrf_num < num)
  {
    // 配列外参照.
    return;
  }

  const float rad = (lrf_offset + num) * angle_increment + angle_min;
  // rad…LRFが取得した距離情報の角度情報.

  float sinx, cosx;
  __sincosf(rad, &sinx, &cosx);
  // sin(rad)とcos(rad)を同時取得.

  const float range =  device_ranges[lrf_offset + num];

  device_xy_data[2 * num] = range * cosx;
  device_xy_data[2 * num + 1] = range * sinx;
  // LRFの距離情報をx-y変換.
}

__global__ void make_hough_graph(float *device_xy_data, int *device_hough_list, const float center_x, const float center_y, const int x_num, const int y_num, const float x_wid, const float y_wid, const float x_range, const float y_range, const float radius){
  // x-yデータからハフ変換したデータ列を作る.

  const int i = blockIdx.x, j = threadIdx.x;
  // blockIdx.x…LRFの何個目の距離情報か threadIdx.x…何個目のxか.

  const float laser_x = device_xy_data[2 * i];
  const float laser_y = device_xy_data[2 * i + 1];
  // LRFの距離情報をx-y変換した情報.

  const bool flag = laser_x > center_x - x_range && laser_x < center_x + x_range && laser_y > center_y - y_range && laser_y < center_y + y_range;
  // LRFの距離情報が注目範囲内に収まっているか確認.
  if (false == flag)
  {
    // 注目範囲外の点だった.
    return;
  }

  const float hough_x = center_x - x_range + j * x_wid;
  // ハフ変換する際のx.

  const float diff_x = (hough_x - laser_x);
  // ハフ変換のxと座標点との差.

  const float eq = radius * radius - diff_x * diff_x;
  // 円としてみた時のyの座標の2乗.


  if (eq < 0)
  {
    // 円として見れない状況.
    return;
  }


  const float root = sqrt(eq);
  const int hough_y_1=(int)(((root+laser_y) - (center_y - y_range)) / y_wid);
  const int hough_y_2=(int)(((-root+laser_y) - (center_y - y_range)) / y_wid);
  // 円の方程式に基づいてハフ変換する際のyを計算.
  // 平方根を取るときに中身が負でないことの確認をしている.

  bool flag1 = hough_y_1 >= 0 && hough_y_1 < y_num;
  bool flag2 = hough_y_2 >= 0 && hough_y_2 < y_num;
  // ハフ変換で求めたyが範囲内にあるかどうか.

  if (flag1)
  {
    device_hough_list[j*y_num+hough_y_1]++;
  }
  if (flag2)
  {
    device_hough_list[j*y_num+hough_y_2]++;
  }
  // ハフ変換のグラフ完成.
}

static void print_hough_gragh(thrust::device_vector<int> &device_hough_list, int x_num, int y_num)
{
  // ハフ変換のグラフを表示.

  thrust::host_vector<int> host_hough_list(x_num * y_num);
  // デバイス用のハフ変換データ格納用のvector.

  thrust::copy(device_hough_list.begin(), device_hough_list.end(), host_hough_list.begin());
  // ホストにハフ変換データをコピー.

  for (int i = 0; i < x_num; i++)
  {
    printf("x=%d:", i);
      for (int j = 0; j < y_num; j++)
      {
        if (device_hough_list[i * y_num + j] >= 2)
        {
          printf("y=%d,%d ", j,  (int)device_hough_list[i*y_num + j]);
        }
      }
    printf("\n");
  }
}

__global__ void find_circle(int *device_hough_list, float *device_circle, int *device_head, const int near_x, const int near_y,const int x_num,const int y_num, const int thr, const float weight){
  // ハフ変換のデータから円を検出する.

  const int i = blockIdx.x, j = threadIdx.x;
  // blockIdx.x…何個目のxか threadIdx.x…何個目のyか.

  const int pos = i * y_num + j;
  // ハフ変換グラフの位置に該当する配列の番号.

  const int val = device_hough_list[pos];
  // 注目する箇所の値.

  if(val < thr)
  {
    // 閾値未満.
    return;
  }

  const int x_start  = i - near_x <0 ? 0 : i - near_x;
  const int x_end =  i + near_x >= x_num ? x_num - 1 : i + near_x;
  const int y_start = j - near_y<0 ? 0 : j - near_y;
  const int y_end = j + near_y >= y_num ? y_num - 1 : j + near_y;
  // 範囲情報を元に探索するxの範囲とyの範囲を決定.

  float x_ave = 0, y_ave = 0, count = 0;
  // xとyの重み付き平均を求めようとしている.
  // _aveは重みを付けた総和、countは重みの総和.
  // weighが線形的に見る範囲の数.

  for (int k = x_start; k <= x_end; k++){
    for (int l = y_start; l <= y_end; l++){
      const int com = device_hough_list[k * y_num + l];
      // 比較する場所の値.

      const bool count_flag = (val < com);

      if (true == count_flag)
      {
        return;
      }

      float z = (k-i)*(k-i)+(l-j)*(l-j);
      z = z < weight * weight ? weight * weight - z : 0;
      // 重みを計算.
      x_ave += com * z * k;
      y_ave += com * z * l;
      count += com * z;
    }
  }

  if (0 != count){
    int my_head = device_head[0]++;
    device_circle[my_head * 2] = x_ave/count;
    device_circle[my_head * 2 + 1] = y_ave/count;
  }
  // flag==trueなら書き込み.

}

__global__ void cuda_select_data(const float *device_xy_data, bool *device_IsOnCircleBound, const int lrf_num, const float p, const float q, const float radius, const float rad_err)
{
  // 点が中心(p,q)半径radiusの円から半径誤差rad_err内にあるかどうかを計算する.
  const int num = blockIdx.x * blockDim.x + threadIdx.x;
  if (num < lrf_num)
  {
   const float laser_x = device_xy_data[2 * num], laser_y = device_xy_data[2 * num + 1];
   const bool flag = abs(sqrt((laser_x - p) * (laser_x - p) + (laser_y - q) * (laser_y - q)) - radius) < rad_err;
   device_IsOnCircleBound[num] = flag;
 }
 return;
}

void select_data(const thrust::host_vector<float>& host_xy_data, const thrust::device_vector<float>& device_xy_data, std::vector<float> *select_datas, const int lrf_num, const float p, const float q, const float radius, const float rad_err, const int warp)
{
  // 中心(p,q)半径radiusの円の半径誤差rad_err内に点が存在するかどうかを計算する. 存在した場合の天データをselect_dataに,その個数をselect_data_numに格納する.

  select_datas->clear();

  thrust::device_vector<bool> device_IsOnCircleBound(lrf_num);
  // デバイス用の円上かのデータ格納用のvector.
  thrust::host_vector<bool> host_IsOnCircleBound(lrf_num);
  // ホスト用の円上かのデータ格納用のvector.

  dim3 block_0((int)floor((lrf_num + warp - 1) / warp), 1, 1);
  dim3 thread_0(warp, 1, 1);

  cuda_select_data<<<block_0,thread_0>>>(
    (float*)thrust::raw_pointer_cast(device_xy_data.data()),
    (bool*)thrust::raw_pointer_cast(device_IsOnCircleBound.data()),
    lrf_num, p, q, radius, rad_err);
  // 円上にあるかどうかを計算.

  thrust::copy(device_IsOnCircleBound.begin(), device_IsOnCircleBound.end(), host_IsOnCircleBound.begin());
  // ホストにデータをコピー.

  for (int i = 0; i < lrf_num; i++){
    if (host_IsOnCircleBound[i]){
      select_datas->push_back(host_xy_data[2 * i]);
      select_datas->push_back(host_xy_data[2 * i + 1]);
    }
  }
}

double eva_deno(const double x_a, const double y_a, const double a, const double b)
{
  // 1点に関する評価関数の分母を求める.
  return x_a * x_a + y_a * y_a - 2 * a * x_a - 2 * b * y_a + a * a + b * b;
}

double eva_val(const double deno, const double rad)
{
  // 1点に関する評価関数の値を求める.
  return (deno - rad * rad) * (deno - rad * rad) / deno;
}

double eva_d(const double deno, const double rad, const double x_ab, const double ab)
{
  // 1点に関する評価関数のa,bの偏微分地を求める.
  return (2 * (deno - rad * rad ) * (2 * ab - 2 * x_ab) - (deno - rad * rad) * (deno - rad * rad) * (2 * ab - 2 * x_ab)) / (deno * deno);
}

boost::optional<mle_data> calc_mle(const std::vector<float>& select_datas, const float rad, float x, float y, const float allow_err, const int limit_count)
{
  boost::optional<mle_data> data;
  // 最尤推定のデータ型.
  int count = 0;
  // ニュートン法を行った回数.
  double sq_j;
  // 誤差値の二乗.

  do{
    double j_a = 0, j_b = 0, j_ml = 0;
    // a,bdでの偏微分値と評価関数.
    const int select_data_num = static_cast<int>(select_datas.size()) / 2;
    // 点列の数.

    for(int i = 0; i < select_data_num; i++){
      const double x_a = select_datas[2 * i], y_a = select_datas[2 * i + 1];
      const double deno = eva_deno(x_a, y_a, x, y);
      j_ml += eva_val(deno, rad) / select_data_num;
      j_a += eva_d(deno, rad, x_a, x) / select_data_num;
      j_b += eva_d(deno, rad, y_a, y) / select_data_num;
    }
    // 評価関数の値と偏微分値の計算.

    sq_j = j_a * j_a + j_b * j_b;
    if (sq_j > allow_err * allow_err){
      j_a = j_a / sqrt(sq_j) * allow_err;
      j_b = j_b / sqrt(sq_j) * allow_err;
    }
    // 移動距離を算出.

    if(data)
    {
      // 最尤推定のデータ型に書き込みがあった.
      if(data->j_ml < j_ml)
      {
        // 前の評価関数値が今の計算値より低かった.
        return data;
      }
    }

    const mle_data tmp_mle_data = {x, y, j_ml};
    data = tmp_mle_data;
    // 値の退避.

    x -= j_a;
    y -= j_b;
  } while (allow_err * allow_err < sq_j && count++ < limit_count);

  return data;
}

boost::optional<position> mle(const std::vector<float>& select_datas, const float rad, const float x, const float y, const float allow_err, const int limit_count)
{
  //最尤推定法を実行
  /*
    select_datas…点列情報
    rad…円の半径
    x_esti/y_esti…求めた座標を格納する先
    allow_err…許容誤差
  */

  boost::optional<position> position_data;

  boost::optional<mle_data> data = calc_mle(select_datas, rad, x, y, allow_err, limit_count);
  // 最尤推定により計算.
  if(!data)
  {
    // 計算できず.
    return position_data;
  }

  /*
  const double len = sqrt(x * x + y * y);
  // 円までの長さ.
  const double pro_x = x + x / len * rad, pro_y  = y + y / len * rad;
  // 今回の円検出特有の円の手前の谷にハマる現象対策.

  double j_ml = 0;
  const int select_data_num = static_cast<int>(select_datas.size()) / 2;
  for (int i = 0; i < select_data_num; i++){
    const double x_a = select_datas[2 * i], y_a = select_datas[2 * i + 1];
    const double deno = eva_deno(x_a, y_a, pro_x, pro_y) / select_data_num;
    j_ml += eva_val(deno, rad) / select_data_num;
  }
  if (data->j_ml > j_ml){
   data = calc_mle(select_datas, rad, pro_x, pro_y, allow_err, limit_count);
   // 最尤推定により計算.

   if(!data)
   {
     // 計算できず.
     return position_data;
   }
  }
  */
  const int select_data_num = static_cast<int>(select_datas.size()) / 2;
  for (int i = 0; i < 100; i++)
  {
    const double random_x = x + random_val(0.15), random_y = y + random_val(0.15);
    double j_ml = 0;
    for (int j = 0; j < select_data_num; j++){
      const double x_a = select_datas[2 * j], y_a = select_datas[2 * j + 1];
      const double deno = eva_deno(x_a, y_a, random_x, random_y) / select_data_num;
      j_ml += eva_val(deno, rad) / select_data_num;
    }
    //printf("j_ml:%f\n",j_ml);
    //printf("dj_ml:%f\n",data->j_ml);
    if (data->j_ml > j_ml){
      data = calc_mle(select_datas, rad, random_x, random_y, allow_err, limit_count);
      // 最尤推定により計算.
      //printf("iizo\n");
      if(!data)
      {
        // 計算できず.
        return position_data;
      }
    }
  }

  const position tmp_position = {data->x, data->y};
  position_data = tmp_position;
  // データの移動

  return position_data;
}

double random_val(double wid){
    return ((double)rand() + 1.0) / ((double)RAND_MAX + 2.0) * 2 * wid - wid;
}
