#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdbool.h>
#include <math.h>
#include <math_functions.h>
#include <time.h>
#include <sys/time.h>
#include <vector>
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/sort.h>
#include "../include/detect_cercle_cuda_p.hpp"

__global__ void cuda_select_data(float *xy_data_d,bool *data_list_d,const int lrf_num,const float p,const float q,const float radius,const float rad_err){
  const int num = blockIdx.x*blockDim.x+threadIdx.x;
  if(num<lrf_num){
   const float laser_x=xy_data_d[2*num],laser_y=xy_data_d[2*num+1];
   const bool flag=abs(sqrt((laser_x-p)*(laser_x-p)+(laser_y-q)*(laser_y-q))-radius)<rad_err;
   data_list_d[num]=flag;
 }
 return;
}

void select_data(float *xy_data,float *xy_data_d, float *select_data,int *select_data_num,const int lrf_num,const float p,const float q,const float radius,const float rad_err,const int warp){
  bool *data_list,*data_list_d;
  data_list=(bool*)malloc(lrf_num*sizeof(bool));
  hipMalloc((void**)&data_list_d,lrf_num*sizeof(bool));
  dim3 block_0((int)floor((lrf_num+warp-1)/warp),1,1);
  dim3 thread_0(warp,1,1);
  cuda_select_data<<<block_0,thread_0>>>(xy_data_d,data_list_d,lrf_num,p,q,radius,rad_err);
  hipMemcpy(data_list,data_list_d,lrf_num*sizeof(bool),hipMemcpyDeviceToHost);
  for(int i=0;i<lrf_num;i++){
   if(data_list[i]){
     int my_head = (*select_data_num)++;
     select_data[my_head*2]=xy_data[2*i];
     select_data[my_head*2+1]=xy_data[2*i+1];
     //printf("selectdata:%d\n",select_data_num_d[0]);
   }
 }
}

void calc_mle(float *select_datas,int select_data_num,const float rad,float *x_esti,float *y_esti,const float allow_err,const int limit_count,float *j_ml){
  float err=0,x_posi[2],y_posi[2],dif;
  x_posi[0]=*x_esti;
  y_posi[0]=*y_esti;
  int count=0;
  do{
    //printf("num:%d\n",select_data_num);
    float j_a=0,j_b=0;
    *j_ml=0;
    for(int i=0;i<select_data_num;i++){
      const float x_a=select_datas[2*i],y_a=select_datas[2*i+1];
      const float val=x_a*x_a+y_a*y_a-2*(*x_esti)*x_a-2*(*y_esti)*y_a+(*x_esti)*(*x_esti)+(*y_esti)*(*y_esti);
      *j_ml+=(val-rad*rad)*(val-rad*rad)/val;
      j_a+=(2*(val-rad*rad)*(2*(*x_esti)-2*x_a)-(val-rad*rad)*(val-rad*rad)*(2*(*x_esti)-2*x_a))/(val*val);
      j_b+=(2*(val-rad*rad)*(2*(*y_esti)-2*y_a)-(val-rad*rad)*(val-rad*rad)*(2*(*y_esti)-2*y_a))/(val*val);
    }
    err=j_a*j_a+j_b*j_b;
    if(err>allow_err){
      j_a=j_a/sqrt(err)*allow_err;
      j_b=j_b/sqrt(err)*allow_err;
    }
    x_posi[1]=x_posi[0];
    y_posi[1]=y_posi[0];
    x_posi[0]=*x_esti;
    y_posi[0]=*y_esti;
    *x_esti-=j_a;
    *y_esti-=j_b;
    //printf("mle x:%f,y:%f,j_a:%f,j_b:%f\n",*x_esti,*y_esti,j_a,j_b);
    dif=(x_posi[1]-*x_esti)*(x_posi[1]-*x_esti)+(y_posi[1]-*y_esti)*(y_posi[1]-*y_esti);
  }while(allow_err*allow_err<err&&count++<limit_count&&!(dif<allow_err*allow_err&&count>1));
}

void mle(float *select_datas,int select_data_num,const float rad,float *x_esti,float *y_esti,const float allow_err,const int limit_count){
  //最尤推定法を実行
  /*
    select_datas…点列情報
    select_data_num…点列の個数
    rad…円の半径
    x_esti/y_esti…求めた座標を格納する先
    allow_err…許容誤差
   */

  float j_ml;
  calc_mle(select_datas,select_data_num,rad,x_esti,y_esti,allow_err,limit_count,&j_ml);
  float len=sqrt((*x_esti)*(*x_esti)+(*y_esti)*(*y_esti));
  float pro_x=(*x_esti)+(*x_esti)/len*rad,pro_y=(*y_esti)+(*y_esti)/len*rad;

  float j_ml2=0;
  for(int i=0;i<select_data_num;i++){
    const float x_a=select_datas[2*i],y_a=select_datas[2*i+1];
    const float val=x_a*x_a+y_a*y_a-2*(*x_esti)*x_a-2*(*y_esti)*y_a+(*x_esti)*(*x_esti)+(*y_esti)*(*y_esti);
    j_ml2+=(val-rad*rad)*(val-rad*rad)/val;
  }
  if(j_ml>j_ml2){
   *x_esti=pro_x;
   *y_esti=pro_y;
   calc_mle(select_datas,select_data_num,rad,x_esti,y_esti,allow_err,limit_count,&j_ml);
  }
}

int detect_cercle_cuda(const std::vector<float>& host_ranges, const int lrf_num, const float angle_min, const float angle_increment, const hough_param_str *hough_param, const int thr2, float *p, float *q, const float rad, const float rad_err1, const float rad_err2, const float rad_err3, const float allow_err1, const float allow_err2, const int warp, const int limit_count){
  /*
    struct timeval start,end;
    時間計測用
    以下使い方
    gettimeofday(&start,NULL);
    hogehoge;
    gettimeofday(&end,NULL);
    printf("time:%d[ns]\n", end.tv_usec - start.tv_usec);

    float elapsed_time_ms=0.0f;
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
    hoge<<<,>>>()
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time_ms,start,stop);
    printf("time:%8.2f ms\n",elapsed_time_ms);
    cudaEventDestory(start);
    cudaEventDestory(end);
  */

  int calc_flag = -1;

  if (lrf_num > 1024)
  {
    std::cout << "too many number to see." << std::endl;
    return calc_flag;
  }
  //gpuの可能並列化数を超えていたらreturn

  const size_t ranges_num = host_ranges.size();
  if (ranges_num < lrf_num)
  {
    // LRFの距離データの数が少なすぎた時.
    std::cout << "few number of lrf data." << std::endl;
    return calc_flag;
  }

  thrust::device_vector<float> device_ranges(host_ranges.size());
  // デバイス用の距離データ格納用のvector.
  thrust::device_vector<float> device_xy_data(2 * lrf_num);
  // デバイス用のx-yデータ格納用のvector.
  thrust::host_vector<float> host_xy_data(2 * lrf_num);
  // ホスト用のx-yデータ格納用のvector.
  thrust::copy(host_ranges.begin(), host_ranges.end(), device_ranges.begin());
  // デバイスに距離情報をコピー.

  dim3 block_0((int)floor((lrf_num+warp-1)/warp),1,1);
  dim3 thread_0(warp,1,1);
  // 並列化の大きさを設定.

  const int lrf_offset = (ranges_num - lrf_num) / 2;
  // lrfデータの中央部分のみを見るため,そのオフセットを指定.

  make_xy_data<<<block_0,thread_0>>>(
    (float *)thrust::raw_pointer_cast(device_ranges.data()),
    (float *) thrust::raw_pointer_cast(device_xy_data.data()),
    lrf_offset, lrf_num, angle_min, angle_increment);
  // x-yデータを作成.

  thrust::copy(device_xy_data.begin(), device_xy_data.end(), host_xy_data.begin());
  // ホストにx-yデータをコピー.

  float *xy_data, *xy_data_d;
  xy_data = (float *) thrust::raw_pointer_cast(host_xy_data.data());
  xy_data_d = (float *) thrust::raw_pointer_cast(device_xy_data.data());

  const float x_range = hough_param->x_num / 2 * hough_param->x_wid;
  const float y_range = hough_param->y_num / 2 * hough_param->y_wid;
  // ハフ変換時に見る範囲.

  thrust::device_vector<int> device_hough_list(hough_param->x_num * hough_param->y_num, 0);
  // デバイス用のハフ変換データ格納用のvector.

  dim3 blocks_1(lrf_num, 1, 1);
  dim3 threads_1(hough_param->x_num,1,1);
  // 並列化.

  make_hough_graph<<<blocks_1,threads_1>>>((float *)thrust::raw_pointer_cast(device_xy_data.data()),
  (int *)thrust::raw_pointer_cast(device_hough_list.data()),
  *p, *q,
  hough_param->x_num, hough_param->y_num,
  hough_param->x_wid, hough_param->y_wid,
  x_range, y_range, rad);

  // print_hough_gragh(device_hough_list, x_num, y_num);
  // ハフ変換グラフを表示.

  thrust::device_vector<float> device_cercle(2 * hough_param->x_num * hough_param->y_num);
  // デバイス用の検出円格納用のvector.
  thrust::host_vector<float> host_cercle(2 * hough_param->x_num * hough_param->y_num);
  // ホスト用の検出円格納用のvector.

  thrust::device_vector<int> device_head(1);
  // デバイス用の検出円数格納用のvector.
  thrust::host_vector<int> host_head(1);
  // ホスト用の検出円数格納用のvector.

  host_head[0] = 0;

  thrust::copy(host_head.begin(), host_head.end(), device_head.begin());
  // デバイスに検出円数をコピー.

  dim3 blocks_2(hough_param->x_num, 1, 1);
  dim3 threads_2(hough_param->y_num, 1,1);

  find_cercle<<<blocks_2,threads_2>>>((int *)thrust::raw_pointer_cast(device_hough_list.data()),
  (float *)thrust::raw_pointer_cast(device_cercle.data()),
  (int *)thrust::raw_pointer_cast(device_head.data()),
  hough_param->near_x, hough_param->near_y,
  hough_param->x_num, hough_param->y_num,
  hough_param->thr, hough_param->weight);
  // 円検出.

  thrust::copy(device_cercle.begin(), device_cercle.end(), host_cercle.begin());
  // ホストにx-yデータをコピー.
  thrust::copy(device_head.begin(), device_head.end(), host_head.begin());
  // ホストにx-yデータをコピー.


  float x_esti, y_esti;
  if(host_head[0] != 0){
    const float x_b = *p, y_b = *q;
    // x-yの中心位置(オフセット).
    x_esti = (float)(host_cercle[0] - hough_param->x_num / 2) * hough_param->x_wid + x_b;
    y_esti = (float)(host_cercle[1] - hough_param->y_num / 2) * hough_param->y_wid + y_b;
    // 検出したx-yデータ.

    float diff = (x_b-x_esti) * (x_b-x_esti) + (y_b-y_esti) * (y_b-y_esti);
    // 中心位置からのズレ.

    for (int i = 1; i < host_head[0]; i++){
      const float now_x = (float)(host_cercle[2 * i] - hough_param->x_num / 2) * hough_param->x_wid + x_b;
      const float now_y = (float)(host_cercle[2 * i + 1] - hough_param->y_num / 2) * hough_param->y_wid + y_b;
      const float now_diff = (x_b-now_x) * (x_b-now_x) + (y_b-now_y) * (y_b-now_y);
      if (diff > now_diff)
      {
        x_esti = now_x;
	      y_esti = now_y;
	      diff = now_diff;
      }
    }
  }
  // 予想の位置に一番近いものを選んでいる.

  /*
  if (host_head[0] != 0){
    printf("%d\n",host_head[0]);
    *p = x_esti;
    *q = y_esti;
    calc_flag = 0;
    return calc_flag;
  }
  */

  if (host_head[0] != 0){
    //ここから2回目の処理
    float *select_datas;
    select_datas=(float*)malloc(lrf_num*2*sizeof(float));
    //選別したlrfのデータを保管

    int select_data_num=0;

    select_data(xy_data,xy_data_d,select_datas,&select_data_num,lrf_num,x_esti,y_esti,rad,rad_err1,warp);
    //lrfデータを選別

    //printf("select:%d\n",select_data_num);
    mle(select_datas,select_data_num,rad,&x_esti,&y_esti,allow_err1,limit_count);
    //最尤推定を実行
    select_data_num=0;
    select_data(xy_data,xy_data_d,select_datas,&select_data_num,lrf_num,x_esti,y_esti,rad,rad_err2,warp);

    //lrfデータを選別
    mle(select_datas,select_data_num,rad,&x_esti,&y_esti,allow_err2,limit_count);
    //二回目の最尤推定を実行

    select_data_num=0;
    select_data(xy_data,xy_data_d,select_datas,&select_data_num,lrf_num,x_esti,y_esti,rad,rad_err3,warp);
    //printf("select:%d\n",select_data_num);
     if(select_data_num>thr2){
      *p = x_esti;
      *q = y_esti;
      calc_flag = 0;
      // printf("p=%f\nq=%f\n",*p,*q);
     }
    free(select_datas);
    }

  //printf("time:%f[ms]\n", (float)(end - start)/CLOCKS_PER_SEC*1000);
  //printf("\n");
  return calc_flag;

}

__global__ void make_xy_data(const float *device_ranges, float *device_xy_data, const int lrf_offset, const int lrf_num, const float angle_min, const float angle_increment)
{
  // 距離データをx-yデータに変換する.

  const int num = blockIdx.x * blockDim.x + threadIdx.x;
  // 見るべき配列の値.

  if (lrf_num < num)
  {
    // 配列外参照.
    return;
  }

  const float rad = (lrf_offset + num) * angle_increment + angle_min;
  // rad…LRFが取得した距離情報の角度情報.

  float sinx, cosx;
  __sincosf(rad, &sinx, &cosx);
  // sin(rad)とcos(rad)を同時取得.

  const float range =  device_ranges[lrf_offset + num];

  device_xy_data[2 * num] = range * cosx;
  device_xy_data[2 * num + 1] = range * sinx;
  // LRFの距離情報をx-y変換.
}

__global__ void make_hough_graph(float *device_xy_data, int *device_hough_list, const float center_x, const float center_y, const int x_num, const int y_num, const float x_wid, const float y_wid, const float x_range, const float y_range, const float radius){
  // x-yデータからハフ変換したデータ列を作る.

  const int i = blockIdx.x, j = threadIdx.x;
  // blockIdx.x…LRFの何個目の距離情報か threadIdx.x…何個目のxか.

  const float laser_x = device_xy_data[2 * i];
  const float laser_y = device_xy_data[2 * i + 1];
  // LRFの距離情報をx-y変換した情報.

  const bool flag = laser_x > center_x - x_range && laser_x < center_x + x_range && laser_y > center_y - y_range && laser_y < center_y + y_range;
  // LRFの距離情報が注目範囲内に収まっているか確認.
  if (false == flag)
  {
    // 注目範囲外の点だった.
    return;
  }

  const float hough_x = center_x - x_range + j * x_wid;
  // ハフ変換する際のx.

  const float diff_x = (hough_x - laser_x);
  // ハフ変換のxと座標点との差.

  const float eq = radius * radius - diff_x * diff_x;
  // 円としてみた時のyの座標の2乗.


  if (eq < 0)
  {
    // 円として見れない状況.
    return;
  }


  const float root = sqrt(eq);
  const int hough_y_1=(int)(((root+laser_y) - (center_y - y_range)) / y_wid);
  const int hough_y_2=(int)(((-root+laser_y) - (center_y - y_range)) / y_wid);
  // 円の方程式に基づいてハフ変換する際のyを計算.
  // 平方根を取るときに中身が負でないことの確認をしている.

  bool flag1 = hough_y_1 >= 0 && hough_y_1 < y_num;
  bool flag2 = hough_y_2 >= 0 && hough_y_2 < y_num;
  // ハフ変換で求めたyが範囲内にあるかどうか.

  if (flag1)
  {
    device_hough_list[j*y_num+hough_y_1]++;
  }
  if (flag2)
  {
    device_hough_list[j*y_num+hough_y_2]++;
  }
  // ハフ変換のグラフ完成.
}

static void print_hough_gragh(thrust::device_vector<int> &device_hough_list, int x_num, int y_num)
{
  // ハフ変換のグラフを表示.

  thrust::host_vector<int> host_hough_list(x_num * y_num);
  // デバイス用のハフ変換データ格納用のvector.

  thrust::copy(device_hough_list.begin(), device_hough_list.end(), host_hough_list.begin());
  // ホストにハフ変換データをコピー.

  for (int i = 0; i < x_num; i++)
  {
    printf("x=%d:", i);
      for (int j = 0; j < y_num; j++)
      {
        if (device_hough_list[i * y_num + j] >= 2)
        {
          printf("y=%d,%d ", j,  (int)device_hough_list[i*y_num + j]);
        }
      }
    printf("\n");
  }
}

__global__ void find_cercle(int *device_hough_list, float *device_cercle, int *device_head, const int near_x,const int near_y,const int x_num,const int y_num, const int thr, const float weight){
  // ハフ変換のデータから円を検出する.

  const int i = blockIdx.x, j = threadIdx.x;
  // blockIdx.x…何個目のxか threadIdx.x…何個目のyか.

  const int pos = i * y_num + j;
  // ハフ変換グラフの位置に該当する配列の番号.

  const int val = device_hough_list[pos];
  // 注目する箇所の値.

  if(val < thr)
  {
    // 閾値未満.
    return;
  }

  const int x_start  = i - near_x <0 ? 0 : i - near_x;
  const int x_end =  i + near_x >= x_num ? x_num - 1 : i + near_x;
  const int y_start = j - near_y<0 ? 0 : j - near_y;
  const int y_end = j + near_y >= y_num ? y_num - 1 : j + near_y;
  // 範囲情報を元に探索するxの範囲とyの範囲を決定.

  float x_ave = 0, y_ave = 0, count = 0;
  // xとyの重み付き平均を求めようとしている.
  // _aveは重みを付けた総和、countは重みの総和.
  // weighが線形的に見る範囲の数.

  for (int k = x_start; k <= x_end; k++){
    for (int l = y_start; l <= y_end; l++){
      const int com = device_hough_list[k * y_num + l];
      // 比較する場所の値.

      const bool count_flag = (val < com);

      if (true == count_flag)
      {
        return;
      }

      float z = (k-i)*(k-i)+(l-j)*(l-j);
      z = z < weight * weight ? weight * weight - z : 0;
      // 重みを計算.
      x_ave += com * z * k;
      y_ave += com * z * l;
      count += com * z;
    }
  }

  if (0 != count){
    int my_head = device_head[0]++;
    device_cercle[my_head * 2] = x_ave/count;
    device_cercle[my_head * 2 + 1] = y_ave/count;
  }
  // flag==trueなら書き込み.

}
