#include "hip/hip_runtime.h"
#include "my_def.h"
#include "detect_cercle_cuda.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdbool.h>
#include <math.h>
#include <math_functions.h>
#include <time.h>
#include <sys/time.h>
__global__ void make_xy_data(float *ranges_d,float *xy_data_d,const int lrf_begin,const int lrf_num,const float angle_min,const float angle_increment){
  const int num = blockIdx.x*blockDim.x+threadIdx.x;
  //printf("%d\n",num);
  if(num<lrf_num){
   const float rad = (lrf_begin + num)*angle_increment + angle_min;
   //rad…LRFが取得した距離情報の角度情報
 
   float sinx, cosx;
   __sincosf(rad, &sinx, &cosx);
   //sin(rad)とcos(rad)を同時取得
 
   const float laser_x = ranges_d[num] * cosx;
   const float laser_y = ranges_d[num] * sinx;
   //LRFの距離情報をx-y変換

   xy_data_d[2*num]=laser_x;
   xy_data_d[2*num+1]=laser_y;
  }
 return;
}
		
__global__ void make_graph(float *xy_data_d, int *list_d, const int lrf_begin, const float angle_min, const float angle_increment,const float x,const float y,const int x_num,const int y_num,const float x_wid,const float y_wid,const float radius){
  const int i = blockIdx.x, j = threadIdx.x;
  //blockIdx.x…LRFの何個目の距離情報か threadIdx.x…何個目のxか

  const float rad = (lrf_begin + i)*angle_increment + angle_min;
  //rad…LRFが取得した距離情報の角度情報
  
  const float laser_x = xy_data_d[2*i];
  const float laser_y = xy_data_d[2*i+1];
  //LRFの距離情報をx-y変換
  
  const float x_range=x_num/2*x_wid+rad;
  const float y_range=y_num/2*y_wid+rad;
  const bool flag=laser_x>x-x_range&&laser_x<x+x_range&&laser_y>y-y_range&&laser_y<y+y_range;
  //LRFの距離情報が注目範囲内に収まっているか確認
  
  if(flag){
    //ブロックごとにある点の距離データを割り当て
    const float hough_x=x-x_num/2*x_wid+j*x_wid;
    //ハフ変換する際のx
    
    const float eq=radius*radius-(hough_x-laser_x)*(hough_x-laser_x);
    const float root=eq>=0?sqrt(eq):0;
    const int hough_y_1=(int)(((root+laser_y)-y+y_num/2*y_wid)/y_wid);
    const int hough_y_2=(int)(((-root+laser_y)-y+y_num/2*y_wid)/y_wid);
    //円の方程式に基づいてハフ変換する際のyを計算
    //平方根を取るときに中身が負でないことの確認をしている
    
    bool flag1=hough_y_1>=0&&hough_y_1<y_num&&eq>=0;
    bool flag2=hough_y_2>=0&&hough_y_2<y_num&&eq>=0;
    //ハフ変換で求めたyが範囲内にあるかどうか
    if(flag1)list_d[j*y_num+hough_y_1]+=1;
    if(flag2)list_d[j*y_num+hough_y_2]+=1;
    //p-q分布の完成
  }
  return;
}

__global__ void find_cercle(int *list_d, float *cercle_d, int *head_d, const int near_x,const int near_y,const int x_num,const int y_num, const int thr, const float weight){
  const int i = blockIdx.x, j = threadIdx.x;
  //blockIdx.x…何個目のxか threadIdx.x…何個目のyか
  
  const int pos = i*y_num + j;
  //p-qグラフの位置に該当する配列の番号
  
  const int val = list_d[pos];
  //注目する箇所の値
  
  const int x_start  = i - near_x <0 ? 0 : i - near_x;
  const int x_end =  i + near_x >= x_num ? x_num - 1 : i + near_x;
  const int y_start = j - near_y<0 ? 0 : j - near_y;
  const int y_end = j + near_y >= y_num ? y_num - 1 : j + near_y;
  //範囲情報を元に探索するxの範囲とyの範囲を決定
  float x_ave=0,x_count=0,y_ave=0,y_count=0;
  //xとyの重み付き平均を求めようとしている
  //_aveは重みを付けた総和、_countは重みの総和
  //weighが線形的に見る範囲の数

  int count = 0;
  for (int k = x_start; k <= x_end; k++){
    for (int l = y_start; l <= y_end; l++){
      const int com=list_d[k*y_num+l];
      const bool count_flag=(val < com)||(val==com&&pos<k*y_num+l);
      count +=(int)count_flag;
      float z=sqrt((float)((k-i)*(k-i)+(l-j)*(l-j)));
      z=z<weight?weight-z:0;
      //重みを計算
      x_ave+=com*z*k;
      x_count+=com*z;
      y_ave+=com*z*l;
      y_count+=com*z;
    }
  }
  
  bool flag = count==0;
  //極大の時true、それ以外flase
  
  flag = flag&&list_d[pos]>thr;
  //閾値を考慮
  
  if (flag){
    int my_head = head_d[0]++;
    cercle_d[my_head * 2] = x_ave/x_count;
    cercle_d[my_head * 2 + 1] = y_ave/y_count;
  }
  //flag==trueなら書き込み

}

__global__ void cuda_select_data(float *xy_data_d,bool *data_list_d,const int lrf_num,const float p,const float q,const float radius,const float rad_err){
  const int num = blockIdx.x*blockDim.x+threadIdx.x;
  if(num<lrf_num){
   const float laser_x=xy_data_d[2*num],laser_y=xy_data_d[2*num+1];
   const bool flag=abs(sqrt((laser_x-p)*(laser_x-p)+(laser_y-q)*(laser_y-q))-radius)<rad_err;
   data_list_d[num]=flag;
 }
 return;
}

void select_data(float *xy_data,float *xy_data_d, float *select_data,int *select_data_num,const int lrf_num,const float p,const float q,const float radius,const float rad_err,const int warp){
  bool *data_list,*data_list_d;
  data_list=(bool*)malloc(lrf_num*sizeof(bool));
  hipMalloc((void**)&data_list_d,lrf_num*sizeof(bool));
  dim3 block_0((int)floor((lrf_num+warp-1)/warp),1,1);
  dim3 thread_0(warp,1,1);
  cuda_select_data<<<block_0,thread_0>>>(xy_data_d,data_list_d,lrf_num,p,q,radius,rad_err);
  hipMemcpy(data_list,data_list_d,lrf_num*sizeof(bool),hipMemcpyDeviceToHost);
  for(int i=0;i<lrf_num;i++){
   if(data_list[i]){
     int my_head = (*select_data_num)++;
     select_data[my_head*2]=xy_data[2*i];
     select_data[my_head*2+1]=xy_data[2*i+1];
     //printf("selectdata:%d\n",select_data_num_d[0]);
   }
 }
}

void calc_mle(float *select_datas,int select_data_num,const float rad,float *x_esti,float *y_esti,const float allow_err,const int limit_count,float *j_ml){
  float err=0,x_posi[2],y_posi[2],dif;
  x_posi[0]=*x_esti;
  y_posi[0]=*y_esti;
  int count=0;
  do{
    //printf("num:%d\n",select_data_num);
    float j_a=0,j_b=0;
    *j_ml=0;
    for(int i=0;i<select_data_num;i++){
      const float x_a=select_datas[2*i],y_a=select_datas[2*i+1];
      const float val=x_a*x_a+y_a*y_a-2*(*x_esti)*x_a-2*(*y_esti)*y_a+(*x_esti)*(*x_esti)+(*y_esti)*(*y_esti);
      *j_ml+=(val-rad*rad)*(val-rad*rad)/val;
      j_a+=(2*(val-rad*rad)*(2*(*x_esti)-2*x_a)-(val-rad*rad)*(val-rad*rad)*(2*(*x_esti)-2*x_a))/(val*val);
      j_b+=(2*(val-rad*rad)*(2*(*y_esti)-2*y_a)-(val-rad*rad)*(val-rad*rad)*(2*(*y_esti)-2*y_a))/(val*val);
    }
    err=j_a*j_a+j_b*j_b;
    if(err>allow_err){
      j_a=j_a/sqrt(err)*allow_err;
      j_b=j_b/sqrt(err)*allow_err;
    }
    x_posi[1]=x_posi[0];
    y_posi[1]=y_posi[0];
    x_posi[0]=*x_esti;
    y_posi[0]=*y_esti;
    *x_esti-=j_a;
    *y_esti-=j_b;
    //printf("mle x:%f,y:%f,j_a:%f,j_b:%f\n",*x_esti,*y_esti,j_a,j_b);
    dif=(x_posi[1]-*x_esti)*(x_posi[1]-*x_esti)+(y_posi[1]-*y_esti)*(y_posi[1]-*y_esti);
  }while(allow_err*allow_err<err&&count++<limit_count&&!(dif<allow_err*allow_err&&count>1));
}

void mle(float *select_datas,int select_data_num,const float rad,float *x_esti,float *y_esti,const float allow_err,const int limit_count){
  //最尤推定法を実行
  /*
    select_datas…点列情報
    select_data_num…点列の個数
    rad…円の半径
    x_esti/y_esti…求めた座標を格納する先
    allow_err…許容誤差
   */

  float j_ml;
  calc_mle(select_datas,select_data_num,rad,x_esti,y_esti,allow_err,limit_count,&j_ml);
  float len=sqrt((*x_esti)*(*x_esti)+(*y_esti)*(*y_esti));
  float pro_x=(*x_esti)+(*x_esti)/len*rad,pro_y=(*y_esti)+(*y_esti)/len*rad;
  
  float j_ml2=0;
  for(int i=0;i<select_data_num;i++){
    const float x_a=select_datas[2*i],y_a=select_datas[2*i+1];
    const float val=x_a*x_a+y_a*y_a-2*(*x_esti)*x_a-2*(*y_esti)*y_a+(*x_esti)*(*x_esti)+(*y_esti)*(*y_esti);
    j_ml2+=(val-rad*rad)*(val-rad*rad)/val;
  }
  if(j_ml>j_ml2){
   *x_esti=pro_x;
   *y_esti=pro_y;
   calc_mle(select_datas,select_data_num,rad,x_esti,y_esti,allow_err,limit_count,&j_ml);
  }
}

void detect_cercle_cuda(float *ranges,const int lrf_num,const int lrf_begin,const float angle_min,const float angle_increment,const float x_wid,const float y_wid,const int x_num,const int y_num,const int near_x,const int near_y,const int thr,const int thr2,float *p,float *q,bool *calc_flag,const float rad,const float rad_err1,const float rad_err2,const float rad_err3,const float allow_err1,const float allow_err2,const float weight, const int warp,const int limit_count){
  /*
    struct timeval start,end;
    時間計測用
    以下使い方
    gettimeofday(&start,NULL);
    hogehoge;
    gettimeofday(&end,NULL);
    printf("time:%d[ns]\n", end.tv_usec - start.tv_usec);

    float elapsed_time_ms=0.0f;
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
    hoge<<<,>>>()
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time_ms,start,stop);
    printf("time:%8.2f ms\n",elapsed_time_ms);
    cudaEventDestory(start);
    cudaEventDestory(end);
  */
  //printf("num:%d,theta_num:%d,rho_num:%d\n", num, theta_num, rho_num);
  *calc_flag=false;

  if (lrf_num>1024){
   printf("%dreturn\n",lrf_num);
   return;
  }
  //gpuの可能並列化数を超えていたらreturn

  float *ranges_d;
  hipMalloc((void**)&ranges_d, lrf_num*sizeof(float));
  hipMemcpy(ranges_d, ranges, lrf_num*sizeof(float), hipMemcpyHostToDevice);
  //device用のLRF距離情報

  float *xy_data,*xy_data_d;
  xy_data=(float*)malloc(2*lrf_num*sizeof(float));
  hipMalloc((void**)&xy_data_d,2*lrf_num*sizeof(float));
  
  dim3 block_0((int)floor((lrf_num+warp-1)/warp),1,1);
  dim3 thread_0(warp,1,1);

  make_xy_data<<<block_0,thread_0>>>(ranges_d,xy_data_d,lrf_begin,lrf_num,angle_min,angle_increment);

  hipMemcpy(xy_data,xy_data_d,2*lrf_num*sizeof(float),hipMemcpyDeviceToHost);
  int *list;
  list = (int*)malloc(x_num*y_num*sizeof(int));
  if (list == NULL)return;
  //p-qグラフ

  for (int i = 0; i < x_num*y_num; i++)list[i] = 0;
  //p-qグラフ初期化


  int *list_d;
  hipMalloc((void**)&list_d, x_num*y_num*sizeof(int));
  hipMemcpy(list_d, list, x_num*y_num * sizeof(int), hipMemcpyHostToDevice);

  //デバイス用p-qグラフ
  
  dim3 blocks_1(lrf_num, 1, 1);
  dim3 threads_1(x_num,1,1);
  //並列化

  make_graph<<<blocks_1,threads_1>>>(xy_data_d,list_d,lrf_begin,angle_min,angle_increment,*p,*q,x_num,y_num,x_wid,y_wid,rad);

  //p-qグラフ作成
  /*
    //p-qグラフを表示
    hipMemcpy(list, list_d, x_num*y_num * sizeof(int), hipMemcpyDeviceToHost);
    //list情報をホストに

    for (int i = 0; i < x_num; i++){
    printf("x=%d:", i);
    for (int j = 0; j < y_num; j++){
    if (list[i*y_num + j] >= 2)printf("y=%d,%d ",j, list[i*y_num + j]);
    }
    printf("\n");
    }
  */  


  float *cercle;
  cercle = (float*)malloc(x_num*y_num* 2* sizeof(float));
  //円情報

  /*
  for (int i = 0; i < theta_num*rho_num*2; i++)line[i] = 0;
  //直線情報初期化
  */

  float *cercle_d;
  hipMalloc((void**)&cercle_d, x_num*y_num*2 * sizeof(float));
  hipMemcpy(cercle_d, cercle, x_num*y_num *2* sizeof(float), hipMemcpyHostToDevice);
  //デバイス用直線情報

  int head[1] = { 0 };
  int *head_d;
  hipMalloc((void**)&head_d, sizeof(int));
  hipMemcpy(head_d, head, sizeof(int), hipMemcpyHostToDevice);
  //配列書き込み位置取得用グローバルメモリ上変数

  dim3 blocks_2(x_num, 1, 1);
  dim3 threads_2(y_num, 1,1);

  find_cercle<<<blocks_2,threads_2>>>(list_d,cercle_d,head_d,near_x,near_y,x_num,y_num,thr,weight);

  hipMemcpy(cercle, cercle_d, x_num*y_num*2 * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(head, head_d, sizeof(int), hipMemcpyDeviceToHost);
  
  float *find_cercles;
  find_cercles = (float*)malloc(head[0]*2*sizeof(float));
  memcpy(find_cercles, cercle, head[0]*2*sizeof(float));
  //不必要にでかいcercleを使わず必要数だけ保管するfind_cercle配列を作る

  printf("head:%d\n", head[0]);
  float x_esti,y_esti;
  if(head[0]!=0){
    const float x_b=*p,y_b=*q;
    x_esti=(float)(find_cercles[0]-x_num/2)* x_wid+x_b;
    y_esti=(float)(find_cercles[1]-y_num/2)* y_wid+y_b;
    float diff=(x_b-x_esti)*(x_b-x_esti)+(y_b-y_esti)*(y_b-y_esti);
    for (int i = 1; i < head[0]; i++){
      const float now_x=(float)(find_cercles[2 * i]-x_num/2)* x_wid+x_b;
      const float now_y=(float)(find_cercles[2 * i+1]-y_num/2)* y_wid+y_b;
      const float now_diff=(x_b-now_x)*(x_b-now_x)+(y_b-now_y)*(y_b-now_y);
      if(diff>now_diff){
	x_esti=now_x;
	y_esti=now_y;
	diff=now_diff;
      }
    }
  }
  //予想のp-qに一番近いものを選んでいる

  //for(int i=0;i<head[0];i++)printf("x:%f\ny:%f\n", (float)(find_cercles[2 * i]-x_num/2)* x_wid+x, (float)(find_cercles[2 * i+1]-y_num/2)* y_wid+y);
  //printf("hough-x:%f\nhough-y:%f\n",x_esti,y_esti); 
  //printf("before_esti_x:%f\nbefore_esti_y:%f\n",*p,*q);

  if(head[0]!=0){
    //ここから2回目の処理
    float *select_datas;
    select_datas=(float*)malloc(lrf_num*2*sizeof(float));
    //選別したlrfのデータを保管

    int select_data_num=0;

    select_data(xy_data,xy_data_d,select_datas,&select_data_num,lrf_num,x_esti,y_esti,rad,rad_err1,warp);
    //lrfデータを選別

    printf("select:%d\n",select_data_num);
    mle(select_datas,select_data_num,rad,&x_esti,&y_esti,allow_err1,limit_count);
    //最尤推定を実行
    select_data_num=0;
    select_data(xy_data,xy_data_d,select_datas,&select_data_num,lrf_num,x_esti,y_esti,rad,rad_err2,warp);

    //lrfデータを選別
    mle(select_datas,select_data_num,rad,&x_esti,&y_esti,allow_err2,limit_count);
    //二回目の最尤推定を実行

    select_data_num=0;
    select_data(xy_data,xy_data_d,select_datas,&select_data_num,lrf_num,x_esti,y_esti,rad,rad_err3,warp);
    printf("select:%d\n",select_data_num);
     if(select_data_num>thr2){
      *p=x_esti;
      *q=y_esti;
      *calc_flag=true;
      printf("p=%f\nq=%f\n",*p,*q);
     }
    free(select_datas);
    }

  free(list);
  free(cercle);
  hipFree(ranges_d);
  hipFree(list_d);
  hipFree(cercle_d);
  hipFree(head_d);
  free(find_cercles);
  
  //printf("time:%f[ms]\n", (float)(end - start)/CLOCKS_PER_SEC*1000);
  //printf("\n");
  return;

}
